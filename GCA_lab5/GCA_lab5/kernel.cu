#include "hip/hip_runtime.h"
﻿#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>
#include <time.h>
#include <string>
#include <fstream>
#include <iostream>

#include <chrono>

#define N 2048

//writeToCSV
//help function to write stuff to csv
void writeRecordToFile(std::string filename, std::string fieldOne, std::string fieldTwo, int fieldThree)
{
    std::ofstream file;
    file.open(filename, std::ios_base::app);
    file << fieldOne << "," << fieldTwo << "," << fieldThree << std::endl;
    file.close();
}

__global__ void getMaxReduction(int* A, int* max)
{
    int i = threadIdx.x + blockIdx.x * blockDim.x;
    int j = N;
    for (int n = 0; n < (log2f(N)); n++)
    {
        j = j / 2;
        if (i < j)
        {
            if (A[i] < A[i + j])
            {
                A[i] = A[i + j];
            }
        }
        __syncthreads();
    }
    if (i == 0)
    {
        *max = A[0];
    }
}

__global__ void getMinReduction(int* A, int* min)
{
    int i = threadIdx.x + blockIdx.x * blockDim.x;
    int j = N;
    for (int n = 0; n < (log2f(N)); n++)
    {
        j = j / 2;
        if (i < j)
        {
            if (A[i] > A[i + j])
            {
                A[i] = A[i + j];
            }
        }
        __syncthreads();
    }
    if (i == 0)
    {
        *min = A[0];
    }
}

__global__ void getSumReduction(int* A, int* sum)
{
    int i = threadIdx.x + blockIdx.x * blockDim.x;
    int j = N;
    for (int n = 0; n < (log2f(N)); n++)
    {
        j = j / 2;
        if (i < j)
        {
            A[i] = A[i] + A[i + j];
        }
        __syncthreads();
    }
    if (i == 0)
    {
        *sum = A[0];
    }
}

__global__ void getProdReduction(int* A, int* prod)
{
    int i = threadIdx.x + blockIdx.x * blockDim.x;
    int j = N;
    for (int n = 0; n < (log2f(N)); n++)
    {
        j = j / 2;
        if (i < j)
        {
            A[i] *= A[i + j];
        }
        __syncthreads();
    }
    if (i == 0)
    {
        *prod = A[0];
    }
}

int* getRandomArray(int n)
{
    size_t size = n * sizeof(int);
    int* A = (int*)malloc(size);
    for (int i = 0; i < n; i++)
    {
        A[i] = rand();
    }
    return A;
}

void executeSync(int* res, bool print = true)
{
    //start chrono
    auto startTimeGPU = std::chrono::steady_clock::now();

    int threadsPerBlock = 1024;
    int blocksPerGrid = (N + threadsPerBlock - 1) / threadsPerBlock;

    //workflow for maximum
    int max = 0;
    int* A = getRandomArray(N);
    int* gpuA = NULL;
    hipMalloc((void**)&gpuA, N * sizeof(int));
    int* gpuMax = NULL;
    hipMalloc((void**)&gpuMax, sizeof(int));
    hipMemcpy(gpuA, A, N * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(gpuMax, &max, sizeof(int), hipMemcpyHostToDevice);
    hipEvent_t startMax, stopMax;
    hipEventCreate(&startMax);
    hipEventCreate(&stopMax);
    hipEventRecord(startMax);
    getMaxReduction << <blocksPerGrid, threadsPerBlock >> > (gpuA, gpuMax);
    hipEventRecord(stopMax);
    hipEventSynchronize(stopMax);
    hipMemcpy(&max, gpuMax, sizeof(int), hipMemcpyDeviceToHost);

    //workflow for minimum
    int min = 0;
    int* B = getRandomArray(N);
    int* gpuB = NULL;
    hipMalloc((void**)&gpuB, N * sizeof(int));
    int* gpuMin = NULL;
    hipMalloc((void**)&gpuMin, sizeof(int));
    hipMemcpy(gpuB, B, N * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(gpuMin, &min, sizeof(int), hipMemcpyHostToDevice);
    hipEvent_t startMin, stopMin;
    hipEventCreate(&startMin);
    hipEventCreate(&stopMin);
    hipEventRecord(startMin);
    getMinReduction << <blocksPerGrid, threadsPerBlock >> > (gpuA, gpuMin);
    hipEventRecord(stopMin);
    hipEventSynchronize(stopMin);
    hipMemcpy(&min, gpuMin, sizeof(int), hipMemcpyDeviceToHost);

    //workflow for sum
    int sum = 0;
    int* C = getRandomArray(N);
    int* gpuC = NULL;
    hipMalloc((void**)&gpuC, N * sizeof(int));
    int* gpuSum = NULL;
    hipMalloc((void**)&gpuSum, sizeof(int));
    hipMemcpy(gpuA, A, N * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(gpuSum, &sum, sizeof(int), hipMemcpyHostToDevice);
    hipEvent_t startSum, stopSum;
    hipEventCreate(&startSum);
    hipEventCreate(&stopSum);
    hipEventRecord(startSum);
    getSumReduction << <blocksPerGrid, threadsPerBlock >> > (gpuA, gpuSum);
    hipEventRecord(stopSum);
    hipEventSynchronize(stopSum);   
    hipMemcpy(&sum, gpuSum, sizeof(int), hipMemcpyDeviceToHost);

    //workflow for prod
    int prod = 0;
    int* D = getRandomArray(N);
    int* gpuD = NULL;
    hipMalloc((void**)&gpuD, N * sizeof(int));
    int* gpuProd = NULL;
    hipMalloc((void**)&gpuProd, sizeof(int));
    hipMemcpy(gpuA, A, N * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(gpuProd, &prod, sizeof(int), hipMemcpyHostToDevice);
    hipEvent_t startProd, stopProd;
    hipEventCreate(&startProd);
    hipEventCreate(&stopProd);
    hipEventRecord(startProd);
    getProdReduction << <blocksPerGrid, threadsPerBlock >> > (gpuA, gpuProd);
    hipEventRecord(stopProd);
    hipEventSynchronize(stopProd);
    hipMemcpy(&prod, gpuProd, sizeof(int), hipMemcpyDeviceToHost);

    auto durationGPU = std::chrono::steady_clock::now() - startTimeGPU;
    
    if (print)
    {
        writeRecordToFile("output4.csv", "sync", std::to_string(N), durationGPU.count());
    }

    //write results to result
    res[0] = max;
    res[1] = min;
    res[2] = sum;
    res[3] = prod;

    //free up memory from GPU
    hipFree(gpuA);
    hipFree(gpuB);
    hipFree(gpuC);
    hipFree(gpuD);
    hipFree(gpuMax);
    hipFree(gpuMin);
    hipFree(gpuSum);
    hipFree(gpuProd);
}

void executeAsync(int* res, bool print = true)
{
    auto startTimeGPU = std::chrono::steady_clock::now();

    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start);

    int threadsPerBlock = 1024;
    int blocksPerGrid = (N + threadsPerBlock - 1) / threadsPerBlock;

    // Create CUDA streams
    hipStream_t stream1, stream2, stream3, stream4;
    hipStreamCreate(&stream1);
    hipStreamCreate(&stream2);
    hipStreamCreate(&stream3);
    hipStreamCreate(&stream4);

    //workflow for maximum
    int max = 0;
    int* A = getRandomArray(N);
    int* gpuA = NULL;
    hipMalloc((void**)&gpuA, N * sizeof(int));
    int* gpuMax = NULL;
    hipMalloc((void**)&gpuMax, sizeof(int));
    hipMemcpyAsync(gpuA, A, N * sizeof(int), hipMemcpyHostToDevice, stream1);
    hipMemcpyAsync(gpuMax, &max, sizeof(int), hipMemcpyHostToDevice, stream1);
    getMaxReduction << <blocksPerGrid, threadsPerBlock, 0, stream1 >> > (gpuA, gpuMax);
    hipMemcpyAsync(&max, gpuMax, sizeof(int), hipMemcpyDeviceToHost, stream1);

    //workflow for minimum
    int min = 0;
    int* B = getRandomArray(N);
    int* gpuB = NULL;
    hipMalloc((void**)&gpuB, N * sizeof(int));
    int* gpuMin = NULL;
    hipMalloc((void**)&gpuMin, sizeof(int));
    hipMemcpyAsync(gpuA, A, N * sizeof(int), hipMemcpyHostToDevice, stream2);
    hipMemcpyAsync(gpuMin, &min, sizeof(int), hipMemcpyHostToDevice, stream2);
    getMinReduction << <blocksPerGrid, threadsPerBlock, 0, stream2 >> > (gpuA, gpuMin);
    hipMemcpyAsync(&min, gpuMin, sizeof(int), hipMemcpyDeviceToHost, stream2);

    //workflow for sum
    int sum = 0;
    int* C = getRandomArray(N);
    int* gpuC = NULL;
    hipMalloc((void**)&gpuC, N * sizeof(int));
    int* gpuSum = NULL;
    hipMalloc((void**)&gpuSum, sizeof(int));
    hipMemcpyAsync(gpuA, A, N * sizeof(int), hipMemcpyHostToDevice, stream3);
    hipMemcpyAsync(gpuSum, &sum, sizeof(int), hipMemcpyHostToDevice, stream3);
    getSumReduction << <blocksPerGrid, threadsPerBlock, 0, stream3 >> > (gpuA, gpuSum);
    hipMemcpyAsync(&sum, gpuSum, sizeof(int), hipMemcpyDeviceToHost, stream3);

    //workflow for prod
    int prod = 0;
    int* D = getRandomArray(N);
    int* gpuD = NULL;
    hipMalloc((void**)&gpuD, N * sizeof(int));
    int* gpuProd = NULL;
    hipMalloc((void**)&gpuProd, sizeof(int));
    hipMemcpyAsync(gpuA, A, N * sizeof(int), hipMemcpyHostToDevice, stream4);
    hipMemcpyAsync(gpuProd, &prod, sizeof(int), hipMemcpyHostToDevice, stream4);
    getProdReduction << <blocksPerGrid, threadsPerBlock, 0, stream4 >> > (gpuA, gpuProd);
    hipMemcpyAsync(&prod, gpuProd, sizeof(int), hipMemcpyDeviceToHost, stream4);

    hipEventRecord(stop);
    hipEventSynchronize(stop);
    auto durationGPU = std::chrono::steady_clock::now() - startTimeGPU;

    if (print)
    {
        writeRecordToFile("output4.csv", "async", std::to_string(N), durationGPU.count());
    }

    //write results to result
    res[0] = max;
    res[1] = min;
    res[2] = sum;
    res[3] = prod;

    //free up memory from GPU
    hipFree(gpuA);
    hipFree(gpuB);
    hipFree(gpuC);
    hipFree(gpuD);
    hipFree(gpuMax);
    hipFree(gpuMin);
    hipFree(gpuSum);
    hipFree(gpuProd);

    // Destroy streams
    hipStreamDestroy(stream1);
    hipStreamDestroy(stream2);
    hipStreamDestroy(stream3);
    hipStreamDestroy(stream4);
}


int main()
{
    size_t resSize = 4 * sizeof(int);
    int* res = (int*)malloc(resSize);

    executeSync(res, false);
    executeAsync(res, false);

    for (int i = 0; i < 1000; i++)
    {
        executeSync(res);
        printf("SYNC:\nmax: %d\nmin: %d\nsum: %d\nprod: %d\n", res[0], res[1], res[2], res[3]);
        executeAsync(res);
        printf("ASYNC:\nmax: %d\nmin: %d\nsum: %d\nprod: %d\n", res[0], res[1], res[2], res[3]);
    }
}
